
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define M 3
#define N 3

__global__ void modify(int *matrix)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    int stride=blockDim.x*gridDim.x;

    for(int i=tid;i<M*N;i+=stride)
    {
        int row=i/N;
        

        if(row==1)
        {
            matrix[i]=pow(matrix[i],2);
        }
        else if(row==2)
        {
            matrix[i]=pow(matrix[i],3);
        }
    }
}


int main()
{
    int matrix[M][N];
    int *d_matrix;



    printf("Enter matrix:\n");
    for(int i=0;i<M;i++)
    {
        for(int j=0;j<N;j++)
        {
            scanf("%d",&matrix[i][j]);
        }
    }

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    hipMalloc((void**)&d_matrix,M*N*sizeof(int));

    hipMemcpy(d_matrix,matrix,M*N*sizeof(int),hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((M*N+256-1)/256);

    modify<<<gridDim,blockDim>>>(d_matrix);

    hipMemcpy(matrix,d_matrix,M*N*sizeof(int),hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    printf("result:\n");
     for(int i=0;i<M;i++)
    {
        for(int j=0;j<N;j++)
        {
            printf("%d ",matrix[i][j]);
        }
        printf("\n");
    }
    hipFree(d_matrix);

}