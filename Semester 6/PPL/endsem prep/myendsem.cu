
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ int position(char *str,int *col,int id)
{
    int pos;

    for(int i=0;i<col[id];i++)
    {
        if(str[id]=='+')
            pos+=col[id]+1;
        
        else
            pos+=col[id]+2;
    }
    return pos;
}


__global__ void sparse(char* str, char* res, int *row, int *col, int len)
{
    int id=threadIdx.x;
    if(id<len)
    {
        int pos=position(str,col,id);
        if(str[id]=='+')
        {
            for(int i=pos;i<pos+col[id]+1;i++)
            {
                res[i]=str[id];
            }
        }
        else
        {
            for(int i=pos;i<pos+col[id]+2;i++)
            {
                res[i]=str[id];
            }
        }
        
    }
}


int main()
{
    char str[100],res[100];

    printf("Enter string: ");
    scanf("%s",str);
    int len=strlen(str);

    int row[len],col[len];

    printf("Enter row mat: ");
    for(int i=0;i<len;i++)
    {
        scanf("%d",&row[i]);
    }
    printf("Enter col mat: ");
    for(int i=0;i<len;i++)
    {
        scanf("%d",&col[i]);
    }

    int *d_row,*d_col;
    char *d_str,*d_res;

    hipMalloc((void**)&d_str,len*sizeof(char));
    hipMalloc((void**)&d_res,len*len*sizeof(char));
    hipMalloc((void**)&d_row,len*sizeof(int));
    hipMalloc((void**)&d_col,len*sizeof(int));

    hipMemcpy(d_str,str,len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_res,res,len*len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_col,col,len*sizeof(int),hipMemcpyHostToDevice);

    sparse<<<1,16>>>(d_str,d_res,d_row,d_col,len);

    hipMemcpy(res,d_res,len*len*sizeof(char),hipMemcpyDeviceToHost);

    printf("result:%s",res);

    hipFree(d_str);
    hipFree(d_res);
    hipFree(d_row);
    hipFree(d_col);

}