#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include ""
#include "hip/hip_runtime.h"


__global__ void transpose(int *a, int *t)
{
    int n=threadIdx.x;
    int m=blockIdx.x;
    int size=blockDim.x-1;
    int size1=gridDim.x;

    t[n*size1+m]=a[m*size+n];
}


int main()
{
    int *a,*t,m,n,i,j;
    int *d_a,*d_t;

    printf("Enter the value of m: ");
    scanf("%d",&m);

    printf("Enter the value of n: ");
    scanf("%d",&n);

    int size=m*n*sizeof(int);

    a=(int*)malloc(m*n*sizeof(int));
    t=(int*)malloc(m*n*sizeof(int));

    printf("Enter the matrix:\n");
    for(int i=0;i<m*n;i++)
    {
        scanf("%d",&a[i]);
    }

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_t,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

    transpose<<<m,n>>(d_a,d_t);

    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    printf("result:\n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",t[i*m+j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_t);

}