
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void sinefun(float*radians, float *sine)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;

    sine[id]=sin(radians[id]);
}

int main()
{
    float *radians,*sine;
    float *d_radians,*d_sine;

    radians=(float*)malloc(N*sizeof(float));
    sine=(float*)malloc(N*sizeof(float));

    for(int i=0;i<N;i++)
    {
        radians[i]=float(i*5/2+0.3)*10;
    }

    hipMalloc((void**)&d_radians,N*sizeof(float));
    hipMalloc((void**)&d_sine,N*sizeof(float));

    hipMemcpy(d_radians,radians,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_sine,sine,N*sizeof(float),hipMemcpyHostToDevice);

    sinefun<<<1,N>>>(d_radians,d_sine);

    hipMemcpy(sine,d_sine,N*sizeof(float),hipMemcpyDeviceToHost);

    printf("Resultant:\n");
    for(int i=0;i<N;i++)
    {
        printf("%f ",sine[i]);
    }

    hipFree(d_sine);
    hipFree(d_radians);

    free(radians);
    free(sine);
}