
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<stdlib.h>

__global__ void print3(char *str,char *out,int ip_len,int n)
{
    int id=threadIdx.x;

    int op_len=n*ip_len;
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<ip_len;j++)
        {
             out[id*op_len+i*ip_len+j]=str[j];
        }
    }
}
   

int main()
{
    char str[]="Hello";
    int N=3;
    int len=strlen(str);

    char *out;
    char *d_str,*d_out;
    out=(char*)malloc(N*strlen(str)*sizeof(char));

    hipMalloc((void**)&d_str,strlen(str)*sizeof(char));
    hipMalloc((void**)&d_out,N*strlen(str)*sizeof(char));

    hipMemcpy(d_str,str,strlen(str)*sizeof(char),hipMemcpyHostToDevice);
    
    print3<<<1,N>>>(d_str,d_out,len,N);

   hipMemcpy(out,d_out,N*strlen(str)*sizeof(char),hipMemcpyDeviceToHost);

    printf("Resultant:%s",out);

    hipFree(d_str);
    hipFree(d_out);

    free(out);

}