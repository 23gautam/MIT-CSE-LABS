#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDAcount(char *A, unsigned int *d_count)
{
    int i = threadIdx.x;
    if (A[i] == 'a')
        atomicAdd(d_count, 1);
}

int main()
{
    char A[N];
    char *d_A;
    unsigned int count = 0, *d_count, *result;
    printf("Enter a string: ");
    scanf("%s", A);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void **)&d_A, (strlen(A) + 1) * sizeof(char));
    hipMalloc((void **)&d_count, sizeof(unsigned int));

    hipMemcpy(d_A, A, (strlen(A) + 1) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    CUDAcount<<<1, strlen(A) + 1>>>(d_A, d_count);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    result = (unsigned int *)malloc(sizeof(unsigned int));
    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurrences of 'a': %u\n", *result);

    printf("Time Taken = %f\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_count);
    free(result);

    return 0;
}
