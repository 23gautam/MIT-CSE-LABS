
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 32

__global__ void vecAdd(int* a,int* b,int* c)
{
    int index=blockIdx.x * blockDim.x + threadIdx.x;
    if(index<N)
    {
        c[index]=a[index]+b[index];
    }
    
}


int main()
{
    int *a, *b, *c;
    int *d_a,*d_b,*d_c;
    int size = N*sizeof(int);

    a=(int*)malloc(size);
    b=(int*)malloc(size);
    c=(int*)malloc(size);

    for(int i=0;i<N;i++)
    {
        a[i]=i;
        b[i]=pow(i,2);
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    vecAdd<<<1,N>>>(d_a,d_b,d_c);

    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    printf("Resultant vector(c):\n");
    for(int i=0;i<N;i++)
    {
        printf("%d ",c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}