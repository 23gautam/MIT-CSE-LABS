#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>

__global__ void counts(char *str, char *word, int *count, int strlen, int wordlen) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < strlen) {
        int i = 0;
        while (i < wordlen && str[tid + i] == word[i]) i++;
        if (i == wordlen) {
            atomicAdd(count, 1);
            tid += wordlen;
        }
        else {
            tid++;
        }
    }
}

int main() {
    const int N = 1024; // Maximum string length
    char input[N];
    char word[N];
    int *count = 0, *d_count;
    int result;

    printf("Enter a string: ");
    fgets(input, sizeof(input), stdin);

    printf("Enter the word to search for: ");
    scanf("%s", word);

    int strlen = strlen(input);
    int wordlen = strlen(word);

    hipMalloc((void**)&d_count, sizeof(int));
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    counts<<<1, strlen>>>(input, word, d_count, strlen, wordlen);

    hipMemcpy(&result, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("Total occurrences of '%s' = %d\n", word, result);

    hipFree(d_count);

    return 0;
}