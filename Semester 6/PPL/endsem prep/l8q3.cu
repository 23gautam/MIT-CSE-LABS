
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 32

__global__ void countword(char *str,char *substr, int *count, int len, int sublen)
{
    int tid=threadIdx.x;
    if(tid<=len-sublen)
    {
        int match=1;
        for(int i=0;i<sublen;i++)
        {
            if(str[tid+i] != substr[i])
            {
                match=0;
                break;
            }
        }
        if(match==1)
        {
            atomicAdd(count,1);
        }
    }
}

int main()
{
    char str[N];
    char substr[N];
    int count;

    char *d_str,*d_substr;
    int *d_count;

    printf("Enter the word: ");
    scanf("%s",str);
    int str_len=strlen(str);

    while (getchar() != '\n');

    printf("Enter the word to search: ");
    scanf("%s",substr);
    int sub_len=strlen(substr);

    hipMalloc((void**)&d_str,str_len*sizeof(char));
    hipMalloc((void**)&d_substr,sub_len*sizeof(char));
    hipMalloc((void**)&d_count,sizeof(int));

    hipMemcpy(d_str,str,str_len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_substr,substr,sub_len*sizeof(char),hipMemcpyHostToDevice);

    countword<<<1,str_len>>>(d_str,d_substr,d_count,str_len,sub_len);

    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);

    printf("count: %d",count);

    hipFree(d_str);
    hipFree(d_substr);
    hipFree(d_count);
}