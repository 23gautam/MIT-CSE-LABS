
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 10
#define M 3

__global__ void convolution1D(int *input, int *mask, int *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int half_mask=M/2;

    if(id<N)
    {
        int val=0;
        for(int i=0;i<M;i++)
        {
            int idx=id+i-half_mask;
            if(idx>=0 && idx<N)
            {
                val+=input[idx]*mask[i];
            }
        }
        out[id]=val;
    }
}
int main()
{
    int *input,*mask,*out;
    int *d_input,*d_mask,*d_out;

    input=(int*)malloc(N*sizeof(int));
    mask=(int*)malloc(M*sizeof(int));
    out=(int*)malloc(N*sizeof(int));

    for(int i=0;i<N;i++)
    {
        input[i]=i+1;
        if(i<M)
            mask[i]=1;
    }


    hipMalloc((void**)&d_input,N*sizeof(int));
    hipMalloc((void**)&d_mask,M*sizeof(int));
    hipMalloc((void**)&d_out,N*sizeof(int));

    hipMemcpy(d_input,input,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_mask,mask,M*sizeof(int),hipMemcpyHostToDevice);

    convolution1D<<<1,N>>>(d_input,d_mask,d_out);

    hipMemcpy(out,d_out,N*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant matrix:\n");
    for(int i=0;i<N;i++)
    {
        printf("%d ",out[i]);
    }
        

    hipFree(d_input);
    hipFree(d_mask);
    hipFree(out);

    free(input);
    free(mask);
    free(out);

    return 0;

}