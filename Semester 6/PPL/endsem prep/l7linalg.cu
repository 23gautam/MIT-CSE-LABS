
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

__global__ void linalgebra(int *x,int *y,int alpha)
{
    int id=blockIdx.x * blockDim.x + threadIdx.x;

    y[id]=alpha*x[id]+y[id];
}

int main()
{
    int *x,*y;
    int alpha;
    int *d_x,*d_y;

    x=(int*)malloc(N*sizeof(int));
    y=(int*)malloc(N*sizeof(int));

    alpha=3;
    for(int i=0;i<N;i++)
    {
        x[i]=i*3;
        y[i]=pow(i,2);
    }

    hipMalloc((void**)&d_x,N*sizeof(int));
    hipMalloc((void**)&d_y,N*sizeof(int));
    
    hipMemcpy(d_x,x,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(int),hipMemcpyHostToDevice);

    linalgebra<<<1,N>>>(d_x,d_y,alpha);

    hipMemcpy(y,d_y,N*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant Matrix:\n");
    for(int i=0;i<N;i++)
    {
        printf("%d ",y[i]);
    }

    hipFree(d_x);
    hipFree(d_y);

    free(x);
    free(y);
}