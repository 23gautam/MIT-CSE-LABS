#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void matmul_col(int *a,int *b,int *c,int ha,int wa)
{
    int cidB=threadIdx.x;
    int wb=blockDim.x;
    int sum;
    for(int ridA=0;ridA<ha;ridA++)
    {
        sum=0;
        for(int k=0;k<wa;k++)
        {
            sum+=a[ridA*wa+k]*b[k*wb+cidB];
        }
        c[ridA*wb+cidB]=sum;
    }
}


int main()
{
    int ha,hb,wa,wb;
    printf("Enter the dim of 'a': ");
    scanf("%d%d",&ha,&wa);

    printf("Enter the dim of 'b': ");
    scanf("%d%d",&hb,&wb);

    int a[ha][wa],b[hb][wb];
    int res[ha][wb];

    printf("Enter 'a': ");
    for(int i=0;i<ha;i++)
    {
        for(int j=0;j<wa;j++)
        {
            scanf("%d",&a[i][j]);
        }
    }

    printf("Enter 'b': ");
    for(int i=0;i<hb;i++)
    {
        for(int j=0;j<wb;j++)
        {
            scanf("%d",&b[i][j]);
        }
    }
    
    int *d_a,*d_b,*d_res;

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    hipMalloc((void**)&d_a,ha*wa*sizeof(int));
    hipMalloc((void**)&d_b,hb*wb*sizeof(int));
    hipMalloc((void**)&d_res,ha*wb*sizeof(int));

    hipMemcpy(d_a,a,ha*wa*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,hb*wb*sizeof(int),hipMemcpyHostToDevice);

    matmul_col<<<1,wb>>>(d_a,d_b,d_res,ha,wa);

    hipMemcpy(res,d_res,ha*wb*sizeof(int),hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    printf("Result:\n");
    for(int i=0;i<ha;i++)
    {
        for(int j=0;j<wb;j++)
        {
            printf("%d ",res[i][j]);
        }
        printf("\n");
    }
    printf("Time taken:%f",elapsedTime);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

}