#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void convolution2d(int *mat, int *mask, int *out, int n, int m, int outSize)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < outSize && col < outSize)
    {
        int sum = 0;
        for (int i = 0; i < m; i++)
        {
            for (int j = 0; j < m; j++)
            {
                int input_row = row + i;
                int input_col = col + j;
                sum += mat[input_row * n + input_col] * mask[i * m + j];
            }
        }
        out[row * outSize + col] = sum;
    }
}

int main()
{
    int m, n;
    printf("Enter dim of matrix: ");
    scanf("%d", &n);

    printf("Enter dim of mask: ");
    scanf("%d", &m);

    int matrix[n][n], mask[m][m];

    int *d_mat, *d_mask, *d_out;

    printf("Enter matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            scanf("%d", &matrix[i][j]);
        }
    }

    printf("Enter mask:\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &mask[i][j]);
        }
    }

    int outSize = n - m + 1;
    int output[outSize][outSize];

    hipMalloc((void **)&d_mat, n * n * sizeof(int));
    hipMalloc((void **)&d_out, outSize * outSize * sizeof(int));
    hipMalloc((void **)&d_mask, m * m * sizeof(int));

    hipMemcpy(d_mat, matrix, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, m * m * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((outSize + blockDim.x - 1) / blockDim.x, (outSize + blockDim.y - 1) / blockDim.y);

    convolution2d<<<gridDim, blockDim>>>(d_mat, d_mask, d_out, n, m, outSize);

    hipMemcpy(output, d_out, outSize * outSize * sizeof(int), hipMemcpyDeviceToHost);

    printf("result:\n");
    for (int i = 0; i < outSize; i++)
    {
        for (int j = 0; j < outSize; j++)
        {
            printf("%d ", output[i][j]);
        }
        printf("\n");
    }

    hipFree(d_mat);
    hipFree(d_mask);
    hipFree(d_out);

    return 0;
}
