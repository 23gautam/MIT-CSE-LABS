#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void comp1(int *a,int *res, int m,int n)
{
    int row=blockIdx.y*blockDim.y+threadIdx.y;
    int col=blockIdx.x*blockDim.x+threadIdx.x;
    int no,rem,newno;

    if(row>=0 && row<m && col>=0 && col<n)
    {
        if(row==0 || row==m-1 || col==0 || col==n-1)
        {
            res[row*n+col]= a[row*n+col];
        }
        else
        {
            no=a[row*n+col]+1;
            rem=0;
            newno=0;
            while(no>0)
            {
              rem=no%2;
              newno=newno*10+rem;
              no=no/2;
            }
            res[row*n+col]=newno;
        }
    }
}

int main()
{
    int m,n;
    printf("Enter the dim: ");
    scanf("%d%d",&m,&n);

    int a[m][n];
    int res[m][n];
    int *d_a,*d_res;

    printf("enter the matrix:\n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&a[i][j]);
        }
    }

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);


    hipMalloc((void**)&d_a,m*n*sizeof(int));
    hipMalloc((void**)&d_res,m*n*sizeof(int));

    hipMemcpy(d_a,a,m*n*sizeof(int),hipMemcpyHostToDevice);

    dim3 blockDim(32,32);
    dim3 gridDim((n+blockDim.x-1)/blockDim.x,(m+blockDim.y-1)/blockDim.y);

    comp1<<<gridDim,blockDim>>>(d_a,d_res,m,n);

    hipMemcpy(res,d_res,m*n*sizeof(int),hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    printf("Result:\n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",res[i][j]);
        }
        printf("\n");
    }
    printf("Time Taken:%f",elapsedTime);

    hipFree(d_a);
    hipFree(d_res);

}