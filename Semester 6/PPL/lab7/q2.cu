
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10 // Number of input elements
#define M 3  // Size of the mask array

__global__ void convolution(int *input, int *mask, int *output) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int half_mask = M / 2;

    if (tid < N) {
        int val = 0;
        for (int i = 0; i < M; i++) {
            int idx = tid + i - half_mask;
            if (idx >= 0 && idx < N) {
                val += input[idx] * mask[i];
            }
        }
        output[tid] = val;
    }
}

int main() {
    int input[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int mask[M] = {1, 1, 1}; // Example mask array
    int output[N];

    int *d_input, *d_mask, *d_output;

    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_mask, M * sizeof(int));
    hipMalloc((void**)&d_output, N * sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, M * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    convolution<<<gridSize, blockSize>>>(d_input, d_mask, d_output);

    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant Array after Convolution:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    return 0;
}
