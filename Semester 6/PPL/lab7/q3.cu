
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024 // Size of the vectors

__global__ void vectorScalarAdd(float alpha, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        y[index] = alpha * x[index] + y[index];
    }
}

int main() {
    float *x, *y;
    float alpha = 2.0f; // Scalar value
    float *d_x, *d_y;
    int size = N * sizeof(float);

    // Allocate host memory
    x = (float*)malloc(size);
    y = (float*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        x[i] = i;
        y[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    // Copy data from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Launch kernel
    vectorScalarAdd<<<1, N>>>(alpha, d_x, d_y);

    // Copy result from device to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Print the resultant vector
    printf("Resultant Vector (y):\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(x);
    free(y);

    return 0;
}
