
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

__global__ void vectorAdd(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate host memory
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i*3;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel
    vectorAdd<<<N,256>>>(d_a, d_b, d_c);

    // Copy result from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print the resultant vector
    printf("Resultant Vector (c):\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
