
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ int position(char *str,int *col,int id)
{
    int pos=0;
    for(int i=0;i<id;i++)
    {
        if(str[i]=='+')
            pos+=col[i]+1;
        
        else
            pos+=col[i]+2;
    }
    return pos;
}


__global__ void sparse(char* str, char* res, int *row, int *col, int len)
{
    int id=threadIdx.x;
    if(id<len)
    {
        int pos=position(str,col,id);
        int nextpos=position(str,col,id+1);

        for(int i=pos;i<nextpos;i++)
        {
          res[i]=str[id];
        }

        
    }
}


int main()
{
    char str[100],res[100];

    printf("Enter string: ");
    scanf("%s",str);
    int len=strlen(str);

    int row[len],col[len];

    printf("Enter row mat: ");
    for(int i=0;i<len;i++)
    {
        scanf("%d",&row[i]);
    }
    printf("Enter col mat: ");
    for(int i=0;i<len;i++)
    {
        scanf("%d",&col[i]);
    }

    int *d_row,*d_col;
    char *d_str,*d_res;

    hipMalloc((void**)&d_str,len*sizeof(char));
    hipMalloc((void**)&d_res,len*len*sizeof(char));
    hipMalloc((void**)&d_row,len*sizeof(int));
    hipMalloc((void**)&d_col,len*sizeof(int));

    hipMemcpy(d_str,str,len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_res,res,len*len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_col,col,len*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_row,row,len*sizeof(int),hipMemcpyHostToDevice);

    sparse<<<1,16>>>(d_str,d_res,d_row,d_col,len);

    hipMemcpy(res,d_res,len*len*sizeof(char),hipMemcpyDeviceToHost);

    printf("result:%s",res);

    hipFree(d_str);
    hipFree(d_res);
    hipFree(d_row);
    hipFree(d_col);

}